
#include "hip/hip_runtime.h"

#include <chrono>

int time_ms() {
  using namespace std::chrono;
  return duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
}

__global__ void cuda_vecs(float* A, float* B, float* C, int N) {
    int id = threadIdx.x;

    if (id < N) {
        C[id] = A[id] + B[id];
    }

}

int add_vecs(float* a, float* b, float* c, int N, int R) {

    int bytes = N * sizeof(float);

    // gpu vectors
    float* d_a;
    float* d_b;
    float* d_c;

    // allocate gpu memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // copy memory to gpu
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);

    // create make block size and count
    int block_size = 320;
    int block_count = N/block_size;

    // call the kernel
    int profile = time_ms();
    for (int i = 0; i < R; i++) {
        cuda_vecs<<<block_size, block_count>>>(d_a, d_b, d_c, N);
    }
    profile = time_ms() - profile;

    // get the results back
    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost);

    // free the gpu memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return profile;
}