
#include "hip/hip_runtime.h"

#include <chrono>
#include <stdlib.h>
#include <iostream>


int NUMBER = 320*1000;
int REPEAT = 10000;


int time_ms() {
  using namespace std::chrono;
  return duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
}

__global__ void cuda_vecs(float* A, float* B, float* C, int N) {
    int id = threadIdx.x;

    if (id < N) {
        C[id] = A[id] + B[id];
    }

}

int add_vecs(float* a, float* b, float* c, int N, int R) {

    int bytes = N * sizeof(float);

    // gpu vectors
    float* d_a;
    float* d_b;
    float* d_c;

    // allocate gpu memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // copy memory to gpu
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);

    // create make block size and count
    int block_size = 320;
    int block_count = N/block_size;

    // call the kernel
    int profile = time_ms();
    for (int i = 0; i < R; i++) {
        cuda_vecs<<<block_size, block_count>>>(d_a, d_b, d_c, N);
    }
    profile = time_ms() - profile;
 
    // get the results back
    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost);

    // free the gpu memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return profile;
}


void cuda_profile() {
    float* a = new float[NUMBER];
    float* b = new float[NUMBER];
    float* c = new float[NUMBER];

    for (int i=0; i<NUMBER; i++) {
        a[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        b[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        c[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }

    int profile = time_ms();
    for (int i=0; i < REPEAT; i++) {
        for (int n=0; n < NUMBER; n++) {
            c[n] = a[n] + b[n];
        }
    }
    profile = time_ms() - profile;
    std::cout << "CPU Time: " << profile << " ms" << std::endl;

    profile = add_vecs(a, b, c, NUMBER, REPEAT);
    std::cout << "CUDA Time: " << profile << " ms" << std::endl;

    delete[] a;
    delete[] b;
    delete[] c;
}