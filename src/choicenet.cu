
#include "choicenet.h"
#include <iostream>
#include <assert.h>

ChoiceNet::ChoiceNet() {
    Synapse* d = new Synapse();

    hipError_t rval = hipMalloc(&gpu_d, sizeof(Synapse));
    assert(rval == hipSuccess);
    rval = hipMemcpy(gpu_d, d, sizeof(Synapse), hipMemcpyHostToDevice);
    assert(rval == hipSuccess);
    delete d;
}

ChoiceNet::~ChoiceNet() {
    hipFree(gpu_d);
}

float ChoiceNet::update(float x_in) {
    //update_Synapses(gpu_d, x_in, 1);
    Synapse* d_out = new Synapse();
    hipError_t rval = hipMemcpy(d_out, gpu_d, sizeof(Synapse), hipMemcpyDeviceToHost);
    assert(rval == hipSuccess);
    float Y = d_out->Y;
    delete d_out;
    return Y;
}