
#include "choicenet.h"
#include <iostream>
#include <assert.h>

ChoiceNet::ChoiceNet() {
    Dendrite* d = new Dendrite();

    hipError_t rval = hipMalloc(&gpu_d, sizeof(Dendrite));
    assert(rval == hipSuccess);
    rval = hipMemcpy(gpu_d, d, sizeof(Dendrite), hipMemcpyHostToDevice);
    assert(rval == hipSuccess);
    delete d;
}

ChoiceNet::~ChoiceNet() {
    hipFree(gpu_d);
}

float ChoiceNet::update(float x_in) {
    //update_Dendrites(gpu_d, x_in, 1);
    Dendrite* d_out = new Dendrite();
    hipError_t rval = hipMemcpy(d_out, gpu_d, sizeof(Dendrite), hipMemcpyDeviceToHost);
    assert(rval == hipSuccess);
    float Y = d_out->Y;
    delete d_out;
    return Y;
}